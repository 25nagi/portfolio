#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <windows.h> // Windows APIの時間関数にアクセスするために必要

#define Grid_x 1024
#define Grid_y 1024
#define Block_x 16
#define Block_y 8

__global__ void thread_num(unsigned long long int *device_result, unsigned long long int cycle, unsigned long long int thread_size);

/* timer */
int timer(void){
    time_t now = time(NULL);
    struct tm *pnow = localtime(&now);
    char buff[128] = "";
    sprintf(buff, "%d:%d:%d", pnow->tm_hour, pnow->tm_min, pnow->tm_sec);
    printf("%s\n", buff);
    return 0;
}

int main(int argc, char **argv){
    if (argc < 2) {
        printf("Usage: %s <number>\n", argv[0]);
        return 1;
    }

    unsigned long long int num = atoll(argv[1]);
    unsigned long long int thread_size = Grid_x * Grid_y * Block_x * Block_y;
    unsigned long long int cycle = num / thread_size + (num % thread_size > 0);
    unsigned long long int *host_result = (unsigned long long int *)malloc(thread_size * sizeof(unsigned long long int));
    unsigned long long int *device_result;
    checkCudaErrors(hipMalloc((void **)&device_result, thread_size * sizeof(unsigned long long int)));

    FILE *csvfile = fopen("prime_times.csv", "w");
    if (csvfile == NULL) {
        printf("Cannot open csv file\n");
        return 1;
    }
    fprintf(csvfile, "Index,Prime,Time,OriginalTime\n");

    LARGE_INTEGER start_time, current_time, frequency;
    QueryPerformanceFrequency(&frequency);
    QueryPerformanceCounter(&start_time);

    unsigned long long int primeIndex = 0;
    long long first_prime_time_ms = -1; // 最初の素数の時間を格納する変数を初期化
    long long elapsed_ms = 0; // 経過時間

    for (unsigned long long int i = 0; i < cycle; i++) {
        thread_num<<<Grid_x, dim3(Block_x, Block_y)>>>(device_result, i, thread_size);
        hipDeviceSynchronize();

        checkCudaErrors(hipMemcpy(host_result, device_result, thread_size * sizeof(unsigned long long int), hipMemcpyDeviceToHost));

        for (unsigned long long int j = 0; j < thread_size; j++) {
            if (host_result[j] != 0) {
                primeIndex++;
                QueryPerformanceCounter(&current_time);
                long long original_time_ms = ((current_time.QuadPart - start_time.QuadPart) * 1000) / frequency.QuadPart;
                // 最初の素数の時間が未設定なら設定する
                if (first_prime_time_ms == -1) {
                    first_prime_time_ms = original_time_ms;
                }
                elapsed_ms = original_time_ms - first_prime_time_ms;
                fprintf(csvfile, "%llu,%llu,%lld,%lld\n", primeIndex, host_result[j], elapsed_ms, original_time_ms);
            }
        }
    }

    fclose(csvfile);
    free(host_result);
    hipFree(device_result);

    return 0;
}

__global__ void thread_num(unsigned long long int *device_result, unsigned long long int cycle, unsigned long long int thread_size){
    unsigned long long int thread_idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned long long int thread_idy = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned long long int thread_idz = threadIdx.z + blockDim.z * blockIdx.z;
    unsigned long long int global_thread_id = thread_idx + Grid_x * Block_x * (thread_idy + Grid_y * Block_y * thread_idz);
    unsigned long long int num_to_check = global_thread_id + cycle * thread_size;
    int flag = 0;

    if (num_to_check < 2) {
        device_result[global_thread_id] = 0;
    } else if (num_to_check == 2) {
        device_result[global_thread_id] = 2;
    } else if (num_to_check % 2 == 0) {
        device_result[global_thread_id] = 0;
    } else {
        unsigned long long int dev = 3;
        while ((dev * dev) <= num_to_check) {
            if (num_to_check % dev == 0) {
                flag = 1;
                break;
            }
            dev += 2;
        }
        if (flag == 0) {
            device_result[global_thread_id] = num_to_check;
        } else {
            device_result[global_thread_id] = 0;
        }
    }
}
